#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/random/uniform_int_distribution.h>
#include <thrust/random/linear_congruential_engine.h>
#include <cstdint>
#include <sys/mman.h>
#include <fcntl.h>
#include <unistd.h>
#include <time.h>
#include <sys/types.h>
#include <stdio.h>
#include <sys/time.h>
#define HISTNUM 1000

// Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError()                                             \
    {                                                                \
        hipError_t e = hipGetLastError();                          \
        if (e != hipSuccess)                                        \
        {                                                            \
            printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, \
                   hipGetErrorString(e));                           \
            exit(EXIT_FAILURE);                                      \
        }                                                            \
    }

/*return a random offset in [0,range). if returning -1, it means the range is not bigger than 0.*/
__device__ int random_offset(int range, int cycle)
{
    if (range <= 0)
    {
        return -1;
    }
    thrust::minstd_rand engine;
    // remove replication
    engine.discard(threadIdx.x * cycle);
    thrust::uniform_int_distribution<> dist(0, range - 1);
    int32_t offset = dist(engine);
    return offset;
}

__global__ void RandomSampling(int *d_indptr, int *d_indice,
                               int *d_src_ids, int *d_dst_ids,
                               int src_num, int sampling_count) // src_num = 232965;
{
    int indexWithinTheGrid = threadIdx.x + blockIdx.x * blockDim.x;
    int gridStride = gridDim.x * blockDim.x;

    for (int i = indexWithinTheGrid; i < src_num; i += gridStride)
    {
        int src_id = d_src_ids[i];
        int start = d_indptr[src_id];
        int end = d_indptr[src_id + 1];
        for (int j = 0; j < sampling_count; j++)
        {
            int offset = random_offset(end - start, j);
            if (offset < 0)
            {
                d_dst_ids[i * sampling_count + j] = -1;
            }
            else
            {
                d_dst_ids[i * sampling_count + j] = d_indice[start + offset];
            }
        }
    }
}

// __device__ int random_offset(int range){
//     if(range <= 0){
//         return -1;
//     }
//     thrust::minstd_rand engine;
//     //remove replication
//     engine.discard(threadIdx.x);
//     thrust::uniform_int_distribution<> dist(0, range - 1);
//     int32_t offset = dist(engine);
//     return offset;
// }
// __global__ void RandomSampling(int* d_indptr, int* d_indice, int* d_src_ids, int* d_dst_ids, int src_num, int sampling_count)
// {
// 	for(int32_t thread_idx = threadIdx.x + blockDim.x * blockIdx.x; thread_idx < src_num * sampling_count; thread_idx += gridDim.x * blockDim.x){
//         int src_id = d_src_ids[thread_idx/sampling_count];

//         int neighbor_count = d_indptr[src_id + 1] - d_indptr[src_id];
//         int start_offset = d_indptr[src_id];

//         int offset = random_offset(neighbor_count);
//         if(offset < 0){
//             d_dst_ids[thread_idx] = -1;
//         }else{
//             d_dst_ids[thread_idx] = d_indice[start_offset + offset];
//         }
//     }
// }

// __global__ void Histogram(int* d_dst_ids, int src_num, int sampling_count, int* global_hist){
// 	for(int32_t thread_idx = threadIdx.x + blockDim.x * blockIdx.x; thread_idx < src_num * sampling_count; thread_idx += gridDim.x * blockDim.x){
//         int dst_id = d_dst_ids[thread_idx];
//         if((dst_id / 40) < HISTNUM){
//             atomicAdd(global_hist + dst_id / 40, 1);
//         }
//     }
// }

//  opt1
__global__ void Histogram(int *d_dst_ids, int src_num, int sampling_count, int *global_hist)
{
    __shared__ int shared_hist[HISTNUM];

    for (int i = threadIdx.x; i < HISTNUM; i += blockDim.x)
    {
        shared_hist[i] = 0;
    }
    __syncthreads();

    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < src_num * sampling_count; i += gridDim.x * blockDim.x)
    {
        int dst_id = d_dst_ids[i];
        int bin_idx = dst_id / 40;

        if (bin_idx < HISTNUM)
        {
            atomicAdd(&shared_hist[bin_idx], 1);
        }
    }
    __syncthreads();

    for (int i = threadIdx.x; i < HISTNUM; i += blockDim.x)
    {
        atomicAdd(&global_hist[i], shared_hist[i]);
    }
}

// opt2
//  __global__ void Histogram(int* d_dst_ids, int src_num, int sampling_count, int* global_hist) {
//      __shared__ int shared_hist[HISTNUM];
//      __shared__ int shared_ids[1024];

//     for (int i = threadIdx.x; i < HISTNUM; i += blockDim.x) {
//         shared_hist[i] = 0;
//     }
//     __syncthreads();

//     int idx = threadIdx.x + blockIdx.x * blockDim.x;
//     // 计算输入数组中的数据在GPU内存中的地址

//     // 将对应块内的输入数据读入共享内存
//     if(tid < src_num * sampling_count){
//         shared_ids[threadIdx.x] = d_dst_ids[idx];
//     }
//     else{
//         shared_ids[threadIdx.x] = 0;
//     }

//     // 等待所有线程同步，确保共享内存全部准备好
//     __syncthreads();

//     for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < src_num * sampling_count; i += gridDim.x * blockDim.x) {
//         // int dst_id = d_dst_ids[i];
//         int dst_id = shared_ids[threadIdx.x];
//         int bin_idx = dst_id / 40;

//         if (bin_idx < HISTNUM) {
//             atomicAdd(&shared_hist[bin_idx], 1);
//         }
//     }
//     __syncthreads();

//     for (int i = threadIdx.x; i < HISTNUM; i += blockDim.x) {
//         atomicAdd(&global_hist[i], shared_hist[i]);
//     }
// }

void mmap_read(std::string &file_name, int32_t *ret)
{
    int64_t index = 0;
    int32_t fd = open(file_name.c_str(), O_RDONLY);
    if (fd == -1)
    {
        std::cout << "cannout open file: " << file_name << "\n";
        return;
    }
    int64_t buf_len = lseek(fd, 0, SEEK_END);
    const int32_t *buf = (int32_t *)mmap(NULL, buf_len, PROT_READ, MAP_PRIVATE, fd, 0);
    const int32_t *buf_end = buf + buf_len / sizeof(int32_t);
    int32_t temp;
    while (buf < buf_end)
    {
        temp = *buf;
        ret[index++] = temp;
        buf++;
    }
    close(fd);
    return;
}

int main(int argc, char **argv)
{
    /*load arxiv dataset*/
    int vertex_num = 232965;
    int edge_num = 114615892;
    int src_num = 232965;
    int sampling_count = 1000;

    int *h_src_ids = (int *)malloc(src_num * sizeof(int));
    int *h_dst_ids = (int *)malloc(src_num * sampling_count * sizeof(int));
    int *h_indptr = (int *)malloc((vertex_num + 1) * sizeof(int));
    int *h_indice = (int *)malloc(edge_num * sizeof(int));

    std::string src_id_file = "src_ids"; // make sure these files in your current working directory.
    std::string indptr_file = "indptr";
    std::string indice_file = "indice";

    mmap_read(src_id_file, h_src_ids);
    mmap_read(indptr_file, h_indptr);
    mmap_read(indice_file, h_indice);

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /*initialize graph storage on GPU*/
    int *d_src_ids;
    hipMalloc(&d_src_ids, src_num * sizeof(int));
    int *d_dst_ids;
    hipMalloc(&d_dst_ids, src_num * sampling_count * sizeof(int));
    int *d_indptr;
    hipMalloc(&d_indptr, (vertex_num + 1) * sizeof(int));
    int *d_indice;
    hipMalloc(&d_indice, edge_num * sizeof(int));
    int *global_hist;
    hipMalloc(&global_hist, HISTNUM * sizeof(int));
    hipMemset(global_hist, 0, HISTNUM * sizeof(int));
    cudaCheckError();

    hipMemcpy(d_src_ids, h_src_ids, src_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_indptr, h_indptr, (vertex_num + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_indice, h_indice, edge_num * sizeof(int), hipMemcpyHostToDevice);
    cudaCheckError();

    dim3 block_num(72, 1);
    dim3 thread_num(1024, 1);
    RandomSampling<<<block_num, thread_num>>>(d_indptr, d_indice, d_src_ids, d_dst_ids, src_num, sampling_count);

    hipEventRecord(start, 0);

    Histogram<<<block_num, thread_num>>>(d_dst_ids, src_num, sampling_count, global_hist);

    cudaCheckError();
    /*return result*/
    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    hipDeviceSynchronize();
    // hipMemcpy(h_dst_ids, d_dst_ids, src_num * sampling_count * sizeof(int), hipMemcpyDeviceToHost);
    // for(int i = 0; i < 10; i++){
    //     for(int j = 0; j < 4; j++){
    //         std::cout<<i<<" "<<j<<" "<<h_dst_ids[i * 4 + j]<<"\n";
    //     }
    // }
    int *h_hist = (int *)malloc(HISTNUM * sizeof(int));
    hipMemcpy(h_hist, global_hist, HISTNUM * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < 100; i++)
    {
        std::cout << i << " " << h_hist[i] << "\n";
    }
    std::cout << "time cost: " << time << " ms\n";
}
